
#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void sum_atomic(const float *inp, float *dest, int64_t size)
{
    const int64_t i = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;

    if (i >= size)
    {
        return;
    }

    atomicAdd(dest, inp[i]);
}